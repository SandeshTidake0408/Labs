#include <iostream>
#include <cstdlib> //For functions like rand().
#include <ctime> // For timing (clock()).
#include <hip/hip_runtime.h>   //CUDA runtime API.

void cpuMatrixMultiply(int* A, int* B, int* C, int N)
{
    // Iterates through rows i and columns j.
    // Multiplies row of A with column of B.
    // Stores result in C[i][j], flattened as C[i * N + j].

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j){
            C[i * N + j] = 0;
            for (int k = 0; k < N; ++k){
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}

__global__ void gpuMatrixMultiply(int* A, int* B, int* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        int value = 0;
        for (int k = 0; k < N; ++k)
        {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void gpuMatrixMultiplyLauncher(int* A, int* B, int* C, int N)
{
    int threadsPerBlock = 16;
    dim3 threads(threadsPerBlock, threadsPerBlock); //Declares a block of threads with 2D layout .Each block contains 16 × 16 = 256 threads.
    dim3 blocks(ceil(float(N) / threadsPerBlock), ceil(float(N) / threadsPerBlock)); //Calculate number of blocks required in both x and y to cover N x N matrix.

    gpuMatrixMultiply<<<blocks, threads>>>(A, B, C, N);//Launch kernel and 
    hipDeviceSynchronize(); //wait for GPU to finish.
}

bool isMatrixEqual(int* A, int* B, int N)
{
    for (int i = 0; i < N * N; ++i)
    {
        if (A[i] != B[i])
        {
            return false;
        }
    }
    return true;
}

int main()
{
    int N = 512;
    int *A, *B, *C, *D, *a, *b, *c;
    int size = N * N * sizeof(int);

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);
    D = (int*)malloc(size);

    for (int i = 0; i < N * N; ++i)
    {
        A[i] = rand() % 1000;
        B[i] = rand() % 1000;
    }

    clock_t start, end;

    start = clock();
    cpuMatrixMultiply(A, B, C, N);
    end = clock();
    float timeTakenCPU = ((float)(end - start)) / CLOCKS_PER_SEC;

    hipMalloc(&a, size);
    hipMalloc(&b, size);
    hipMalloc(&c, size);

    hipMemcpy(a, A, size, hipMemcpyHostToDevice);
    hipMemcpy(b, B, size, hipMemcpyHostToDevice);

    start = clock();
    gpuMatrixMultiplyLauncher(a, b, c, N);
    hipMemcpy(D, c, size, hipMemcpyDeviceToHost);
    end = clock();
    float timeTakenGPU = ((float)(end - start)) / CLOCKS_PER_SEC;

    hipFree(a);
    hipFree(b);
    hipFree(c);

    bool success = isMatrixEqual(C, D, N);

    printf("Matrix Multiplication\n");
    printf("--------------------\n");
    printf("CPU Time: %f seconds\n", timeTakenCPU);
    printf("GPU Time: %f seconds\n", timeTakenGPU);
    printf("Speed Up: %f\n", timeTakenCPU / timeTakenGPU);
    printf("Verification: Matrices are %s\n", success ? "equal" : "not equal");

    free(A);
    free(B);
    free(C);
    free(D);

    return 0;
}

